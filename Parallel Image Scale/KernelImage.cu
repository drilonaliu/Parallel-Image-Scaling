#include "hip/hip_runtime.h"
#include "KernelImage.cuh"
#include <iostream>
//#include ""

//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>


/*
* Kernel Method for scaling image using Billinear Interpolation.
* 
* @param image - Uchar array of source image.
* @param scaledImage - uchar array of the scaled image.
* @param m - width scale.
* @param n - height scale.
* @param r - rows of source image.
* @paarm c - coloumns of src image.
*/
__global__ void cudaScaleImage(uchar* image, uchar* scaledImage, int m, int n, int r, int c) {

	int thread_index = threadIdx.x + blockDim.x * blockIdx.x;

	if (thread_index < m * n * r * c) {
		int i = thread_index % (m*r); //column
		int j = thread_index / (c*n); //row

		double a = (i - 1) * (r - 1) / (m * r - 1) + 1;
		double b = (j - 1) * (c - 1) / (n * c - 1) + 1;

		int  ii = round(a);
		int  jj = round(b);

		scaledImage[thread_index] = image[ii + r * jj];
	}
}