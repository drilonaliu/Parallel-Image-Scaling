#include "hip/hip_runtime.h"
#include "ParallelImageFunctions.cuh"
#include "KernelImage.cuh"


/* Parallel Image Scaling Function using Billinear Interpolation
*
* @param image - the image to scale
* @param m - width scale
* @param n - height scale
*
* @return scaled image
*/
Mat parallelScaleImage(Mat image, int m, int n) {


	uchar* imageData = image.data;
	Mat scaledImage(m*image.rows, n*image.cols, CV_8U);

	//Image Sizes
	int imageSize = image.rows * image.cols;
	int scaledImageSize = m * n * imageSize;

	//Device pointers 
	uchar* d_image;
	uchar* d_scaledImage;

	//Memory Allocation
	hipMalloc((void**)&d_image, imageSize * sizeof(uchar));
	hipMalloc((void**)&d_scaledImage, scaledImageSize * sizeof(uchar));

	//Memory Copy
	hipMemcpy(d_image, imageData,imageSize*sizeof(uchar),hipMemcpyHostToDevice);

	//Launch Kernel
	//Launch Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (int)ceil(scaledImageSize / threadsPerBlock);
	dim3 grid_size(scaledImage.rows, scaledImage.cols);
	cudaScaleImage << <blocksPerGrid, threadsPerBlock >> > (d_image, d_scaledImage, m, n, image.rows, image.cols);
	
	//Wait For Cuda
	hipDeviceSynchronize();

	//Copy Results back
	hipMemcpy(scaledImage.data , d_scaledImage, scaledImageSize * sizeof(uchar), hipMemcpyDeviceToHost);

	//Free device memory
	hipFree(d_image);
	hipFree(d_scaledImage);

	return scaledImage;
}
